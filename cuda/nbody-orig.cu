#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Particle;

__global__
void calcForces(Particle *p, float dt, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;

        for (int j = 0; j < N; j++) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        p[i].vx += dt*Fx;
        p[i].vy += dt*Fy;
        p[i].vz += dt*Fz;
    }
}

int main(const int argc, const char** argv) {

    const int
        N = 30000,
        nSteps = 100,
        nBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    const float dt = 0.01f; // time step

    size_t n_bytes = N*sizeof(Particle);
    Particle *particles = (Particle*)malloc(n_bytes);

    for (int i = 0; i < N; i++) {
        Particle *p = particles + i;
        p->x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        p->y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        p->z = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        p->vx = 0;
        p->vy = 0;
        p->vz = 0;
    }

    Particle *d_p;
    hipMalloc(&d_p, n_bytes);

    StartTimer();
    for (int iter = 1; iter <= nSteps; iter++) {

        hipMemcpy(d_p, particles, n_bytes, hipMemcpyHostToDevice);
        calcForces <<<nBlocks, BLOCK_SIZE>>>(d_p, dt, N);
        hipMemcpy(particles, d_p, n_bytes, hipMemcpyDeviceToHost);

        for (int i = 0 ; i < N; i++) { // integrate position
            particles[i].x += particles[i].vx*dt;
            particles[i].y += particles[i].vy*dt;
            particles[i].z += particles[i].vz*dt;
        }
    }

    printf("N=%d, Titer=%0.3f s\n",
        N,
        GetTimer() / nSteps / 1000.0);

    free(particles);
    hipFree(d_p);
}
